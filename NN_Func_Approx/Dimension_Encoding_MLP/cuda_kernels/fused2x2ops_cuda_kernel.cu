#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>

#define BLOCK_DIM 16

template <typename scalar_t>
__device__ inline scalar_t clamp(scalar_t d, scalar_t min, scalar_t max) {
  const scalar_t t = d < min ? min : d;
  return t > max ? max : t;
}

template <typename scalar_t>
__global__ void bilinear2x2_cuda_forward_kernel(
    torch::PackedTensorAccessor32<scalar_t,3,torch::RestrictPtrTraits> input, 
    const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> weights,
    const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> grids, 
    torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> input_buffer, 
    size_t s0, size_t s1, size_t gx, size_t gy, size_t num_layers)
{
    // Each thread computes one batch of 2x2 operation.
    size_t bi = blockIdx.x * blockDim.x + threadIdx.x; // batch_size
    size_t gi = blockIdx.y * blockDim.y + threadIdx.y; // input_dim//2
    // bi == batch index ; gi == group index
    if ((bi >= s0) || (gi >= s1)){
        return;
    }
    
    scalar_t x, y, a00, a01, a10, a11;
    int ix, iy;
    size_t gap, gidx, gidy;

    for (size_t layer_i = 0; layer_i < num_layers; layer_i++)
    {
      gap = 1 << layer_i ;
      gidx = (gi%gap) + (gi / gap)*(1<<(layer_i+1)) ;
      gidy = gidx+gap;
      /// save inputs for backward propagation      
      x = input[bi][gidx];
      y = input[bi][gidy];

      input_buffer[layer_i][bi][gidx] = x;
      input_buffer[layer_i][bi][gidy] = y;

      /// using a00 as temp variable
      a00 = x * weights[layer_i][gi][0][0] + y * weights[layer_i][gi][1][0];
      y = x * weights[layer_i][gi][0][1] + y * weights[layer_i][gi][1][1];
      x = a00;

      /// inputs (x,y) are calculated for range 0,1 globally...
      x = x* (scalar_t)(gx-1); // value on x grid
      y = y* (scalar_t)(gy-1); // value on y grid

      ix = clamp<int>((int)x, 0, gx-2);  // index of x grid
      iy = clamp<int>((int)y, 0, gy-2);  // index of y grid

      x -= (scalar_t) ix; // true value of x,y for the given piece
      y -= (scalar_t) iy; // given piece is in range [0,1]

      a00 = grids[layer_i][gi][0][ix][iy];
      a01 = grids[layer_i][gi][0][ix][iy+1] - a00;
      a10 = grids[layer_i][gi][0][ix+1][iy] - a00;
      a11 = grids[layer_i][gi][0][ix+1][iy+1] - grids[layer_i][gi][0][ix+1][iy] - a01;

      input[bi][gidx] = a00 + x*a10 + y*a01 + x*y*a11 ;

      ////////// for second grid with same pair
      a00 = grids[layer_i][gi][1][ix][iy];
      a01 = grids[layer_i][gi][1][ix][iy+1] - a00;
      a10 = grids[layer_i][gi][1][ix+1][iy] - a00;
      a11 = grids[layer_i][gi][1][ix+1][iy+1] - grids[gi][1][ix+1][iy] - a01;

      input[bi][gidy] = a00 + x*a10 + y*a01 + x*y*a11 ;

      __syncthreads();
    }
    
    return;
}

/// here, we expect the tensor not to be transposed, but does same bmm
std::vector<torch::Tensor> bilinear2x2_cuda_forward(
    torch::Tensor input,
    torch::Tensor weights,
    torch::Tensor grids,) {

  const auto s0 = input.size(0);
  const auto s1 = grids.size(1);

  const auto num_layers = grids.size(0);

  const auto gx = grids.size(3);
  const auto gy = grids.size(4);

  // input has shape -> bs, group*2 == input_dim
  // grids has shape -> layer_idx, group, 2, grid_x, grid_y
  // weights has shape -> layer_idx, group, 2, 2

  // std::cout<<"Batch Size "<<batch_size<<" Input Size "<<input.size(1)<<","<<input.size(2)<<std::endl;
  dim3 threads_per_block(BLOCK_DIM, BLOCK_DIM);
  // spreading batch across multiple blocks and thread
  dim3 blocks_per_grid(1, 1);
  blocks_per_grid.x = std::ceil(static_cast<double>(s0) /
                                static_cast<double>(threads_per_block.x));
  blocks_per_grid.y = std::ceil(static_cast<double>(s1) /
                                static_cast<double>(threads_per_block.y));

  auto input_buffer = torch::zeros({num_layers, s0, s1*2}, input.device());

  AT_DISPATCH_FLOATING_TYPES(input.type(), "bilinear2x2_forward_cuda", ([&] {
    bilinear2x2_cuda_forward_kernel<scalar_t><<<blocks_per_grid, threads_per_block>>>(
        input.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>(),
        weights.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
        grids.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
        input_buffer.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),
        s0, s1, gx, gy, num_layers);
  }));

  return {input, input_buffer};
}















template <typename scalar_t> // mat1 is X, mat2 is W -> Y = X.W
__global__ void bilinear2x2_cuda_backward_kernel(
    const input_buffer.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),
    const weights.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
    const grids.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
    grad_output.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),
    // del_input.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),
    del_weights.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
    del_grids.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
    size_t s0, size_t s1, size_t gx, size_t gy)
{
    // Each thread computes one batch of 2x2 operation.
    size_t bi = blockIdx.x * blockDim.x + threadIdx.x; // batch_size
    size_t gi = blockIdx.y * blockDim.y + threadIdx.y; // input_dim//2
    // bi == batch index ; gi == group index
    if ((bi >= s0) || (gi >= s1)){
        return;
    }

    scalar_t x, y, a00, a01, a10, a11, dy, da01, da10, da11, dinp_x, dinp_y;
    int ix, iy;
    size_t gap, gidx, gidy;

    for (size_t layer_i = num_layers; layer_i >= 0; layer_i--)
    {
      gap = 1 << layer_i ;
      gidx = (gi%gap) + (gi / gap)*(1<<(layer_i+1)) ;
      gidy = gidx+gap;
      /// save inputs for backward propagation      

      x = input_buffer[layer_i][bi][gidx] ;
      y = input_buffer[layer_i][bi][gidy] ;

      /// using a00 as temp variable
      a00 = x * weights[layer_i][gi][0][0] + y * weights[layer_i][gi][1][0];
      y = x * weights[layer_i][gi][0][1] + y * weights[layer_i][gi][1][1];
      x = a00;

      /////////////////////////////////////////

      /// inputs (x,y) are calculated for range 0,1 globally...
      x = x* (scalar_t)(gx-1); // value on x grid
      y = y* (scalar_t)(gy-1); // value on y grid

      ix = clamp<int>((int)x, 0, gx-2);  // index of x grid
      iy = clamp<int>((int)y, 0, gy-2);  // index of y grid

      x -= (scalar_t) ix; // true value of x,y for the given piece
      y -= (scalar_t) iy; // given piece is in range [0,1]

      a00 = grids[layer_i][gi][0][ix][iy];
      a01 = grids[layer_i][gi][0][ix][iy+1] - a00;
      a10 = grids[layer_i][gi][0][ix+1][iy] - a00;
      a11 = grids[layer_i][gi][0][ix+1][iy+1] - grids[layer_i][gi][0][ix+1][iy] - a01;

      // compute del input here
      dy = del_output[bi][gix];

      dinp_x = dy*(a10+y*a11);
      dinp_y = dy*(a01+x*a11);

      da01 = dy*y;
      da10 = dy*x;
      da11 = da10*y;

      del_grids[layer_i][bi][gi][0][ix+1][iy+1] = da11; 
      del_grids[layer_i][bi][gi][0][ix][iy+1] = da01 - da11; 
      del_grids[layer_i][bi][gi][0][ix+1][iy] = da10 - da11;
      del_grids[layer_i][bi][gi][0][ix][iy] = dy - da01 - da10 + da11; 

      //////////// for second pairwise, doing the same


      a00 = grids[gi][1][ix][iy];
      a01 = grids[gi][1][ix][iy+1] - a00;
      a10 = grids[gi][1][ix+1][iy] - a00;
      a11 = grids[gi][1][ix+1][iy+1] - grids[gi][1][ix+1][iy] - a01;

      dy = del_output[bi][gi][1];

      // this is del output for 2x2 linear layer before bilinear
      dinp_x += dy*(a10+y*a11);
      dinp_y += dy*(a01+x*a11);
      dinp_x *= (scalar_t)(gx-1); // correcting for the initial multiplication to gx
      dinp_y *= (scalar_t)(gy-1);

      da01 = dy*y;
      da10 = dy*x;
      da11 = da10*y;

      del_grids[layer_i][bi][gi][1][ix+1][iy+1] = da11; 
      del_grids[layer_i][bi][gi][1][ix][iy+1] = da01 - da11; 
      del_grids[layer_i][bi][gi][1][ix+1][iy] = da10 - da11;
      del_grids[layer_i][bi][gi][1][ix][iy] = dy - da01 - da10 + da11; 

      /////////////////////////////////////////
      /////////////////////////////////////////
      /// for input weight layer
      del_weight[layer_i][bi][gi][0][0] = dinp_x*input[bi][gix];
      del_weight[layer_i][bi][gi][0][1] = dinp_y*input[bi][giy];
      del_weight[layer_i][bi][gi][1][0] = dinp_x*input[bi][gix];
      del_weight[layer_i][bi][gi][1][1] = dinp_y*input[bi][giy];
      
      /// this is actually del_input, however the variables are reused for next iteraton
      grad_output[bi][gix] = dinp_x * weights[layer_i][bi][0][0] + 
                            dinp_y * weights[layer_i][bi][0][1];
      grad_output[bi][giy] = dinp_x * weights[layer_i][bi][1][0] + 
                            dinp_y * weights[layer_i][bi][1][1];

      __syncthreads();
    }
    
    return;
}


std::vector<torch::Tensor> bilinear2x2_cuda_backward(
    torch::Tensor input_buffer,
    torch::Tensor weights,
    torch::Tensor grids,
    torch::Tensor grad_output) {

  // input_buffer has shape -> layer_idx, bs, group*2 == input_dim
  // grids has shape -> layer_idx, group, 2, grid_x, grid_y
  // weights has shape -> layer_idx, group, 2, 2
  // grad_output has shape -> bs, group*2

  const auto s0 = input_buffer.size(1);
  const auto s1 = grids.size(1);

  const auto num_layers = grids.size(0);

  const auto gx = grids.size(3);
  const auto gy = grids.size(4);

  dim3 threads_per_block(BLOCK_DIM, BLOCK_DIM);
  // spreading batch across multiple blocks and thread
  dim3 blocks_per_grid(1, 1);
  blocks_per_grid.x = std::ceil(static_cast<double>(s0) /
                                static_cast<double>(threads_per_block.x));
  blocks_per_grid.y = std::ceil(static_cast<double>(s1) /
                                static_cast<double>(threads_per_block.y));

  auto del_weights = torch::zeros(
                    {num_layers, s0, s1, 2, 2},
                    input_buffer.device());
  auto del_grids = torch::zeros(
                    {num_layers, s0, s1, grids.size(2), gx, gy},
                    input_buffer.device()); // grids.size(2) == 2

  AT_DISPATCH_FLOATING_TYPES(input.type(), "bilinear2x2_backward_cuda", ([&] {
    bilinear2x2_cuda_backward_kernel<scalar_t><<<blocks_per_grid, threads_per_block>>>(
        input_buffer.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),
        grids.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
        grad_output.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),
        del_weights.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
        del_grids.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
        s0, s1, gx, gy);
  }));

  return {grad_output, torch::sum(del_grids, 1)};
}