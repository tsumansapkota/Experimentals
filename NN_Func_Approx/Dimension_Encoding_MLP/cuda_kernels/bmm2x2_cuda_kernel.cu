#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>

#define BLOCK_DIM 16

/*
template <typename scalar_t>
__global__ void bmm2x2_cuda_forward_kernel(
    const scalar_t* __restrict__ mat_1, 
    const scalar_t* __restrict__ mat_2, 
    scalar_t* __restrict__ mat_3, 
    size_t b)
{
    // Each thread computes one batch of 2x2 matmul.
    size_t i4 = blockIdx.x * blockDim.x + threadIdx.x;

    if (i4 >= b){
        return;
    }
    i4 = i4*4 ;
    mat_3[i4] = mat_1[i4] * mat_2[i4] + mat_1[i4+1]*mat_2[i4+2];
    mat_3[i4+1] = mat_1[i4] * mat_2[i4+1] + mat_1[i4+1]*mat_2[i4+3];
    mat_3[i4+2] = mat_1[i4+2] * mat_2[i4] + mat_1[i4+3]*mat_2[i4+2];
    mat_3[i4+3] = mat_1[i4+2] * mat_2[i4+1] + mat_1[i4+3]*mat_2[i4+3];

    return;
}

template <typename scalar_t> // mat1 is X, mat2 is W -> Y = X.W
__global__ void bmm2x2_cuda_backward_kernel(
    const scalar_t* __restrict__ mat_1, 
    const scalar_t* __restrict__ mat_2,
    const scalar_t* __restrict__ d_out, 
    scalar_t* __restrict__ d_mat1, 
    scalar_t* __restrict__ d_mat2, 
    size_t b)
{
    // dmat1 is dX, dmat2 is dW, d_out is dY
    // Each thread computes one batch of 2x2 matmul.
    size_t i4 = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i4 >= b){
        return;
    }
    i4 = i4*4 ;

    /// computing dX = dY.(W^t)
    d_mat1[i4] = d_out[i4] * mat_2[i4] + d_out[i4+1]*mat_2[i4+1];
    d_mat1[i4+1] = d_out[i4] * mat_2[i4+2] + d_out[i4+1]*mat_2[i4+3];
    d_mat1[i4+2] = d_out[i4+2] * mat_2[i4] + d_out[i4+3]*mat_2[i4+1];
    d_mat1[i4+3] = d_out[i4+2] * mat_2[i4+2] + d_out[i4+3]*mat_2[i4+3];

    /// computing dW = dX^t.dY
    d_mat2[i4] = mat_1[i4] * d_out[i4] + mat_1[i4+2]*d_out[i4+2];
    d_mat2[i4+1] = mat_1[i4] * d_out[i4+1] + mat_1[i4+2]*d_out[i4+3];
    d_mat2[i4+2] = mat_1[i4+1] * d_out[i4] + mat_1[i4+3]*d_out[i4+2];
    d_mat2[i4+3] = mat_1[i4+1] * d_out[i4+1] + mat_1[i4+3]*d_out[i4+3];
    return;
}
*/

template <typename scalar_t>
__global__ void bmm2x2_cuda_forward_kernel(
    const torch::PackedTensorAccessor32<scalar_t,3,torch::RestrictPtrTraits> input, 
    const torch::PackedTensorAccessor32<scalar_t,3,torch::RestrictPtrTraits> weight, 
    torch::PackedTensorAccessor32<scalar_t,3,torch::RestrictPtrTraits> output, 
    size_t s0, size_t s1)
{
    // Each thread computes one batch of 2x2 matmul.
    size_t i0 = blockIdx.x * blockDim.x + threadIdx.x; // input_dim//2 
    size_t i1 = blockIdx.y * blockDim.y + threadIdx.y; // batch_size
    if ((i0 >= s0) || (i1 >= s1)){
        return;
    }
    
    output[i0][i1][0] = input[i0][i1][0] * weight[i0][0][0] + 
                          input[i0][i1][1] * weight[i0][1][0];
    output[i0][i1][1] = input[i0][i1][0] * weight[i0][0][1] + 
                          input[i0][i1][1] * weight[i0][1][1];

    return;
}


template <typename scalar_t> // mat1 is X, mat2 is W -> Y = X.W
__global__ void bmm2x2_cuda_backward_kernel(
    const scalar_t* __restrict__ mat_1, 
    const scalar_t* __restrict__ mat_2,
    const scalar_t* __restrict__ d_out, 
    scalar_t* __restrict__ d_mat1, 
    scalar_t* __restrict__ d_mat2, 
    size_t b)
{
    // dmat1 is dX, dmat2 is dW, d_out is dY
    // Each thread computes one batch of 2x2 matmul.
    size_t i4 = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i4 >= b){
        return;
    }
    i4 = i4*4 ;

    /// computing dX = dY.(W^t)
    d_mat1[i4] = d_out[i4] * mat_2[i4] + d_out[i4+1]*mat_2[i4+1];
    d_mat1[i4+1] = d_out[i4] * mat_2[i4+2] + d_out[i4+1]*mat_2[i4+3];
    d_mat1[i4+2] = d_out[i4+2] * mat_2[i4] + d_out[i4+3]*mat_2[i4+1];
    d_mat1[i4+3] = d_out[i4+2] * mat_2[i4+2] + d_out[i4+3]*mat_2[i4+3];

    /// computing dW = dX^t.dY
    d_mat2[i4] = mat_1[i4] * d_out[i4] + mat_1[i4+2]*d_out[i4+2];
    d_mat2[i4+1] = mat_1[i4] * d_out[i4+1] + mat_1[i4+2]*d_out[i4+3];
    d_mat2[i4+2] = mat_1[i4+1] * d_out[i4] + mat_1[i4+3]*d_out[i4+2];
    d_mat2[i4+3] = mat_1[i4+1] * d_out[i4+1] + mat_1[i4+3]*d_out[i4+3];
    return;
}


//////////////////////////////////////////////////

std::vector<torch::Tensor> bmm2x2_cuda_forward(
    torch::Tensor input,
    torch::Tensor weights) {

  const auto s0 = input.size(0);
  const auto s1 = input.size(1);

  // std::cout<<"Batch Size "<<batch_size<<" Input Size "<<input.size(1)<<","<<input.size(2)<<std::endl;
  dim3 threads_per_block(BLOCK_DIM, BLOCK_DIM);
  // spreading batch across multiple blocks and thread
  dim3 blocks_per_grid(1, 1);
  blocks_per_grid.x = std::ceil(static_cast<double>(s0) /
                                static_cast<double>(threads_per_block.x));
  blocks_per_grid.y = std::ceil(static_cast<double>(s1) /
                                static_cast<double>(threads_per_block.y));

  // size_t threads_per_block = BLOCK_DIM*BLOCK_DIM;
  // size_t blocks_per_grid = std::ceil(static_cast<double>(batch_size) /
  //                                 static_cast<double>(threads_per_block));

  // const int threads_per_block = 1024; // default is 1024
  // const dim3 blocks_per_grid((batch_size + threads - 1) / threads, batch_size);

  auto output = torch::zeros_like(input);

  AT_DISPATCH_FLOATING_TYPES(input.type(), "bmm2x2_forward_cuda", ([&] {
    bmm2x2_cuda_forward_kernel<scalar_t><<<blocks_per_grid, threads_per_block>>>(
        input.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),
        weights.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),
        output.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),
        s0, s1);
  }));

  return {output};
}


std::vector<torch::Tensor> bmm2x2_cuda_backward(
    torch::Tensor input,
    torch::Tensor weights,
    torch::Tensor grad_output) {

  const auto batch_size = input.size(0);

  std::cout<<"Batch Size"<<batch_size<<std::endl;

  // dim3 threads_per_block(BLOCK_DIM, BLOCK_DIM);
  // // spreading batch across multiple blocks and thread
  // dim3 blocks_per_grid(1, 1);
  // blocks_per_grid.x = std::ceil(static_cast<double>(p) /
  //                               static_cast<double>(threads_per_block.x));
  // blocks_per_grid.y = std::ceil(static_cast<double>(m) /
  //                               static_cast<double>(threads_per_block.y));


  size_t threads_per_block = BLOCK_DIM*BLOCK_DIM;
  size_t blocks_per_grid = std::ceil(static_cast<double>(batch_size) /
                                  static_cast<double>(threads_per_block));
  // const int threads = 1024;
  // const dim3 blocks((state_size + threads - 1) / threads, batch_size);

  auto del_input = torch::zeros_like(input);
  auto del_weights = torch::zeros_like(input);

  AT_DISPATCH_FLOATING_TYPES(input.type(), "bmm2x2_backward_cuda", ([&] {
    bmm2x2_cuda_backward_kernel<scalar_t><<<blocks_per_grid, threads_per_block>>>(
        input.data<scalar_t>(),
        weights.data<scalar_t>(),
        grad_output.data<scalar_t>(),
        del_input.data<scalar_t>(),
        del_weights.data<scalar_t>(),
        batch_size);
  }));

  return {del_input, del_weights};
}